#include "hip/hip_runtime.h"



__global__ void match_count_kernel(int *res, char** lines, char** patterns, int** dfas, int* pattern_size, int* line_size, int* score_map) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int i = 0, j;
    int m = pattern_size[threadIdx.x], n = line_size[blockIdx.x];
    char *line = lines[blockIdx.x];
    char *pattern = patterns[threadIdx.x];
    int *nxt = dfas[threadIdx.x];

    // i is the pointer of 'line'
    // j is the pointer of 'pattern'
    while (i < n) {

        // start a single search (first set j to 0)
        j = 0;
        while (i < n && j < m) {
            if (j == -1 || line[i] == pattern[j]) {
                i++;
                j++;
            } else {
                j = nxt[j];
            }
        }

        // right after a single search
        // If j == m: we have found one match, so res ++
        if (j == m) {
            res[tid] ++;
            i = i - j + 1;
            continue;
        }

        // Otherwise: we have traversed to the end of 'line', so just break
        break;
    }
}


extern "C" int match_count_multiple(char** lines, char** patterns, int** dfas, int* pattern_size, int* line_size, int* score_map) {

    char **dev_lines, **dev_patterns;
    int** dev_dfas;
    int *dev_pattern_size, *dev_line_size, *dev_score_map, *dev_res;

    /* =============== Malloc memory on GPU =============== */

    // malloc lines
    hipMalloc((void**)&dev_lines, sizeof(char*) * 4);
    for (int k = 0; k < 4; k ++) {
        hipMalloc((void **) &(dev_lines[k]), sizeof(char) * line_size[k]);
    }

    // malloc patterns
    hipMalloc((void**)&dev_patterns, sizeof(char*) * 16);
    for (int k = 0; k < 16; k ++) {
        hipMalloc((void **) &(dev_patterns[k]), sizeof(char) * pattern_size[k]);
    }

    // malloc dfas
    hipMalloc((void**)&dev_dfas, sizeof(int*) * 16);
    for (int k = 0; k < 16; k ++) {
        hipMalloc((void **) &(dev_dfas[k]), sizeof(int) * pattern_size[k]);
    }

    // malloc pattern_size
    hipMalloc((void**)&dev_pattern_size, sizeof(int) * 16);

    // malloc line_size
    hipMalloc((void**)&dev_line_size, sizeof(int) * 4);

    // malloc score_map
    hipMalloc((void**)&dev_score_map, sizeof(int) * 16);

    // malloc result
    hipMalloc((void**)&dev_res, sizeof(int) * 64);


    /* =============== Copy memory from RAM to GPU device =============== */

    hipMemcpy(dev_lines, lines, sizeof(char*) * 4, hipMemcpyHostToDevice);
    for (int k = 0; k < 4; k ++) {
        hipMemcpy(dev_lines[k], lines[k], sizeof(char) * line_size[k], hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_patterns, patterns, sizeof(char*) * 4, hipMemcpyHostToDevice);
    for (int k = 0; k < 4; k ++) {
        hipMemcpy(dev_lines[k], lines[k], sizeof(char) * pattern_size[k], hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_dfas, dfas, sizeof(int*) * 16, hipMemcpyHostToDevice);
    for (int k = 0; k < 16; k ++) {
        hipMemcpy(dev_dfas[k], dfas[k], sizeof(int) * (pattern_size[k] + 1), hipMemcpyHostToDevice);
    }

    hipMemcpy(dev_pattern_size, pattern_size, sizeof(int) * 16, hipMemcpyHostToDevice);
    hipMemcpy(dev_line_size, line_size, sizeof(int) * 4, hipMemcpyHostToDevice);
    hipMemcpy(dev_score_map, score_map, sizeof(int) * 16, hipMemcpyHostToDevice);

    int res[64];
    for (int k = 0; k < 64; k ++) res[k] = 0;
    hipMemcpy(dev_res, res, sizeof(int) * 64, hipMemcpyHostToDevice);

    int out = 0;
    match_count_kernel<<<4, 16>>>(dev_res, dev_lines, dev_patterns, dev_dfas, dev_pattern_size, dev_line_size, dev_score_map);
    hipMemcpy(res, dev_res, sizeof(int) * 64, hipMemcpyDeviceToDevice);

    // reduce process
    for (int k = 0; k < 64; k ++) out += res[k];

    /* =============== Copy memory from RAM to GPU device =============== */

    for (int k = 0; k < 4; k ++) {
        hipFree(dev_lines[k]);
    }

    for (int k = 0; k < 16; k ++) {
        hipFree(dev_patterns[k]);
        hipFree(dev_dfas[k]);
    }

    hipFree(dev_res);
    hipFree(dev_line_size);
    hipFree(dev_pattern_size);
    hipFree(dev_score_map);

    return out;
}