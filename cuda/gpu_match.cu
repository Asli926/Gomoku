#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpu_match.cuh"
#include <stdio.h>

inline void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA runtime error: %s.\n", hipGetErrorString(err));
        fflush(stdout);
    }
}

__global__
void match_count_kernel(int *res, char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int i = 0, j;
    int m = pattern_size[threadIdx.x], n = line_size[blockIdx.x];
    char *line = lines + (20 * blockIdx.x);
    char *pattern = patterns + 6 * threadIdx.x;
    int *nxt = dfas + 7 * threadIdx.x;

    // i is the pointer of 'line'
    // j is the pointer of 'pattern'
    while (i < n) {

        // start a single search (first set j to 0)
        j = 0;
        while (i < n && j < m) {
            if (j == -1 || line[i] == pattern[j]) {
                i++;
                j++;
            } else {
                j = nxt[j];
            }
        }

        // right after a single search
        // If j == m: we have found one match, so res ++
        if (j == m) {
            res[tid] += score_map[threadIdx.x];
            i = i - j + 1;
            continue;
        }

        // Otherwise: we have traversed to the end of 'line', so just break
        break;
    }
}


extern "C"
int match_count_multiple(char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {

    char *dev_lines, *dev_patterns;
    int *dev_dfas;
    int *dev_pattern_size, *dev_line_size, *dev_score_map, *dev_res;

    /* =============== Malloc memory on GPU =============== */

    // malloc lines (4 lines of 20 characters)
    checkCudaError(hipMalloc((void**)&dev_lines, sizeof(char) * 4 * 20));

    // malloc patterns (16 patterns of 6 characters)
    checkCudaError(hipMalloc((void**)&dev_patterns, sizeof(char) * 16 * 6));

    // malloc dfas (16 dfa arrays of 7 integers)
    checkCudaError(hipMalloc((void**)&dev_dfas, sizeof(int) * 16 * 7));

    // malloc pattern_size
    checkCudaError(hipMalloc((void**)&dev_pattern_size, sizeof(int) * 16));

    // malloc line_size
    checkCudaError(hipMalloc((void**)&dev_line_size, sizeof(int) * 4));

    // malloc score_map
    checkCudaError(hipMalloc((void**)&dev_score_map, sizeof(int) * 16));

    // malloc result
    checkCudaError(hipMalloc((void**)&dev_res, sizeof(int) * 64));


    /* =============== Copy memory from RAM to GPU device =============== */

    checkCudaError(hipMemcpy(dev_lines, lines, sizeof(char) * 4 * 20, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dev_patterns, patterns, sizeof(char) * 4 * 6, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dev_dfas, dfas, sizeof(int) * 16 * 7, hipMemcpyHostToDevice));

    checkCudaError(hipMemcpy(dev_pattern_size, pattern_size, sizeof(int) * 16, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dev_line_size, line_size, sizeof(int) * 4, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dev_score_map, score_map, sizeof(int) * 16, hipMemcpyHostToDevice));

    int res[64];
    for (int k = 0; k < 64; k ++) res[k] = 0;
    checkCudaError(hipMemcpy(dev_res, res, sizeof(int) * 64, hipMemcpyHostToDevice));

    int out = 0;
    match_count_kernel<<<4, 16>>>(dev_res, dev_lines, dev_patterns, dev_dfas, dev_pattern_size, dev_line_size, dev_score_map);
    checkCudaError(hipMemcpy(res, dev_res, sizeof(int) * 64, hipMemcpyDeviceToHost));

    // reduce process
    for (int k = 0; k < 64; k ++) out += res[k];
    if (out != 0) {
        printf("out: %d\n", out);
        fflush(stdout);
    }

    /* =============== Copy memory from RAM to GPU device =============== */

    checkCudaError(hipFree(dev_lines));
    checkCudaError(hipFree(dev_patterns));
    checkCudaError(hipFree(dev_dfas));
    checkCudaError(hipFree(dev_res));
    checkCudaError(hipFree(dev_line_size));
    checkCudaError(hipFree(dev_pattern_size));
    checkCudaError(hipFree(dev_score_map));

    return out;
}