#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpu_match.cuh"
#include <stdio.h>
#include <cstdlib>

inline void checkCudaError(hipError_t err, const char* loc) {
    if (err != hipSuccess) {
        printf("[%s]CUDA runtime error: %s.\n", loc, hipGetErrorString(err));
        fflush(stdout);
    }
}

__global__
void match_count_kernel(int *res, char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {
    __shared__ int temp[64];
    int tid = threadIdx.x;
    temp[tid] = 0;

    int i = 0, j;
    int threadId = tid % 16, blockId = tid / 16;
    int m = pattern_size[threadId], n = line_size[blockId];
    char *line = lines + (20 * blockId);
    char *pattern = patterns + 6 * threadId;
    int *nxt = dfas + 7 * threadId;

    // i is the pointer of 'line'
    // j is the pointer of 'pattern'
    while (i < n) {

        // start a single search (first set j to 0)
        j = 0;
        while (i < n && j < m) {
            if (j == -1 || line[i] == pattern[j]) {
                i++;
                j++;
            } else {
                j = nxt[j];
            }
        }

        // right after a single search
        // If j == m: we have found one match, so res ++
        if (j == m) {
            temp[tid] += score_map[threadId];
            i = i - j + 1;
            continue;
        }

        // Otherwise: we have traversed to the end of 'line', so just break
        break;
    }

    __syncthreads(); // synchronize all threads

    if (tid == 0)
    {
        int sum = 0;
        for (int t = 0; t < 64; t++)
        {
            sum += temp[t];
        }
        *res = sum;
    }

}


extern "C"
int match_count_multiple(char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {

    char *dev_lines, *dev_patterns;
    int *dev_dfas;
    int *dev_pattern_size, *dev_line_size, *dev_score_map, *dev_res;

    /* =============== Malloc memory on GPU =============== */

    // malloc lines (4 lines of 20 characters)
    checkCudaError(hipMalloc((void**)&dev_lines, sizeof(char) * 4 * 20), "Malloc lines");

    // malloc patterns (16 patterns of 6 characters)
    checkCudaError(hipMalloc((void**)&dev_patterns, sizeof(char) * 16 * 6), "Malloc patterns");

    // malloc dfas (16 dfa arrays of 7 integers)
    checkCudaError(hipMalloc((void**)&dev_dfas, sizeof(int) * 16 * 7), "Malloc dfas");

    // malloc pattern_size
    checkCudaError(hipMalloc((void**)&dev_pattern_size, sizeof(int) * 16), "Malloc pattern size");

    // malloc line_size
    checkCudaError(hipMalloc((void**)&dev_line_size, sizeof(int) * 4), "Malloc line size");

    // malloc score_map
    checkCudaError(hipMalloc((void**)&dev_score_map, sizeof(int) * 16), "Malloc score map");

    // malloc result
    checkCudaError(hipMalloc((void**)&dev_res, sizeof(int)), "Malloc result");


    /* =============== Copy memory from RAM to GPU device =============== */

    checkCudaError(hipMemcpy(dev_lines, lines, sizeof(char) * 4 * 20, hipMemcpyHostToDevice), "copy lines");
    checkCudaError(hipMemcpy(dev_patterns, patterns, sizeof(char) * 6 * 16, hipMemcpyHostToDevice), "copy patterns");
    checkCudaError(hipMemcpy(dev_dfas, dfas, sizeof(int) * 16 * 7, hipMemcpyHostToDevice), "copy dfas");

    checkCudaError(hipMemcpy(dev_pattern_size, pattern_size, sizeof(int) * 16, hipMemcpyHostToDevice), "copy pattern size");
    checkCudaError(hipMemcpy(dev_line_size, line_size, sizeof(int) * 4, hipMemcpyHostToDevice), "copy line size");
    checkCudaError(hipMemcpy(dev_score_map, score_map, sizeof(int) * 16, hipMemcpyHostToDevice), "copy score map");

    int *res = (int*) malloc(sizeof(int));

    match_count_kernel<<<1, 64>>>(dev_res, dev_lines, dev_patterns, dev_dfas, dev_pattern_size, dev_line_size, dev_score_map);
//    hipDeviceSynchronize();
    checkCudaError(hipMemcpy(res, dev_res, sizeof(int), hipMemcpyDeviceToHost), "copy result from GPU");
    int out = *res;
//    if (out != 0) {
//        printf("out: %d\n", out);
//        fflush(stdout);
//    }

    /* =============== Free memory =============== */

    checkCudaError(hipFree(dev_lines), "free lines");
    checkCudaError(hipFree(dev_patterns), "free patterns");
    checkCudaError(hipFree(dev_dfas), "free dfas");
    checkCudaError(hipFree(dev_res), "free res");
    checkCudaError(hipFree(dev_line_size), "free line size");
    checkCudaError(hipFree(dev_pattern_size), "free pattern size");
    checkCudaError(hipFree(dev_score_map), "free score map");

    free(res);

    return out;
}