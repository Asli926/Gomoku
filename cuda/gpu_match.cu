#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gpu_match.cuh"
#include <stdio.h>
#include <cstdlib>

inline void checkCudaError(hipError_t err, const char* loc) {
    if (err != hipSuccess) {
        printf("[%s]CUDA runtime error: %s.\n", loc, hipGetErrorString(err));
        fflush(stdout);
    }
}

__global__
void match_count_kernel(int *res, char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int i = 0, j;
    int m = pattern_size[threadIdx.x], n = line_size[blockIdx.x];
    char *line = lines + (20 * blockIdx.x);
    char *pattern = patterns + 6 * threadIdx.x;
    int *nxt = dfas + 7 * threadIdx.x;

    // i is the pointer of 'line'
    // j is the pointer of 'pattern'
    while (i < n) {

        // start a single search (first set j to 0)
        j = 0;
        while (i < n && j < m) {
            if (j == -1 || line[i] == pattern[j]) {
                i++;
                j++;
            } else {
                j = nxt[j];
            }
        }

        // right after a single search
        // If j == m: we have found one match, so res ++
        if (j == m) {
            res[tid] += score_map[threadIdx.x];
            i = i - j + 1;
            continue;
        }

        // Otherwise: we have traversed to the end of 'line', so just break
        break;
    }
}


extern "C"
int match_count_multiple(char* lines, char* patterns, int* dfas, int* pattern_size, int* line_size, int* score_map) {

    char *dev_lines, *dev_patterns;
    int *dev_dfas;
    int *dev_pattern_size, *dev_line_size, *dev_score_map, *dev_res;

    /* =============== Malloc memory on GPU =============== */

    // malloc lines (4 lines of 20 characters)
    checkCudaError(hipMalloc((void**)&dev_lines, sizeof(char) * 4 * 20), "Malloc lines");

    // malloc patterns (16 patterns of 6 characters)
    checkCudaError(hipMalloc((void**)&dev_patterns, sizeof(char) * 16 * 6), "Malloc patterns");

    // malloc dfas (16 dfa arrays of 7 integers)
    checkCudaError(hipMalloc((void**)&dev_dfas, sizeof(int) * 16 * 7), "Malloc dfas");

    // malloc pattern_size
    checkCudaError(hipMalloc((void**)&dev_pattern_size, sizeof(int) * 16), "Malloc pattern size");

    // malloc line_size
    checkCudaError(hipMalloc((void**)&dev_line_size, sizeof(int) * 4), "Malloc line size");

    // malloc score_map
    checkCudaError(hipMalloc((void**)&dev_score_map, sizeof(int) * 16), "Malloc score map");

    // malloc result
    checkCudaError(hipMalloc((void**)&dev_res, sizeof(int) * 64), "Malloc result");


    /* =============== Copy memory from RAM to GPU device =============== */

    checkCudaError(hipMemcpy(dev_lines, lines, sizeof(char) * 4 * 20, hipMemcpyHostToDevice), "copy lines");
    checkCudaError(hipMemcpy(dev_patterns, patterns, sizeof(char) * 6 * 16, hipMemcpyHostToDevice), "copy patterns");
    checkCudaError(hipMemcpy(dev_dfas, dfas, sizeof(int) * 16 * 7, hipMemcpyHostToDevice), "copy dfas");

    checkCudaError(hipMemcpy(dev_pattern_size, pattern_size, sizeof(int) * 16, hipMemcpyHostToDevice), "copy pattern size");
    checkCudaError(hipMemcpy(dev_line_size, line_size, sizeof(int) * 4, hipMemcpyHostToDevice), "copy line size");
    checkCudaError(hipMemcpy(dev_score_map, score_map, sizeof(int) * 16, hipMemcpyHostToDevice), "copy score map");

    int *res = malloc(sizeof(int) * 64);
    for (int k = 0; k < 64; k ++) res[k] = 0;
    checkCudaError(hipMemcpy(dev_res, res, sizeof(int) * 64, hipMemcpyHostToDevice), "copy result to GPU");

    int out = 0;
    match_count_kernel<<<4, 16>>>(dev_res, dev_lines, dev_patterns, dev_dfas, dev_pattern_size, dev_line_size, dev_score_map);
    checkCudaError(hipMemcpy(res, dev_res, sizeof(int) * 64, hipMemcpyDeviceToHost), "copy result from GPU");

    // reduce process
    for (int k = 0; k < 64; k ++) out += res[k];
    if (out != 0) {
        printf("out: %d\n", out);
        fflush(stdout);
    }

    /* =============== Free memory =============== */

    checkCudaError(hipFree(dev_lines), "free lines");
    checkCudaError(hipFree(dev_patterns), "free patterns");
    checkCudaError(hipFree(dev_dfas), "free dfas");
    checkCudaError(hipFree(dev_res), "free res");
    checkCudaError(hipFree(dev_line_size), "free line size");
    checkCudaError(hipFree(dev_pattern_size), "free pattern size");
    checkCudaError(hipFree(dev_score_map), "free score map");

    free(res);

    return 0;
}